#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stddef.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include "image.h"
#include "Gaussian.h"
#include "GPUconvolution.h"
#include "GPUMagDir.h"
#include "GPUNonMaxSuppression.h"
#include "GPUHysteresis.h"
#include "GPUEdgeLinking.h"
//#include "image_template.h" not for this assignment C cannot handle the templates



//Can use sigma as a constant to run the code
//#define sigma .6

int main(int argc, char **argv)
{




    struct timeval start_total, end_total, start_image_read, end_image_read, start_edge_write, 
    end_edge_write, start_memCpy, end_memCpy, start_hor, end_hor, start_ver, end_ver, start_mag,
    end_mag, start_dir, end_dir, start_sort, end_sort, start_supp, end_supp, start_edge, end_edge,
    start_hys, end_hys, start_memCpydevice, end_memCpydevice;

  	//gettimeofday(&start_ete, NULL);

    int i, j;
    int height;
    int width;
    int *image;

    float sigma = atof(argv[2]);
    int w = 2*(round(2.5*sigma -.5))+1;

    gettimeofday(&start_image_read, NULL);
        read_image_template(argv[1], &image, &width, &height);
    gettimeofday(&end_image_read, NULL);

	gettimeofday(&start_total, NULL);

    double *gauss;// = (double*)malloc(sizeof(double)*w);
    double *gaussDeriv;// = (double*)malloc(sizeof(double)*w);
    double *flip_gaussDeriv;

    gauss = gaussianKernel(sigma);
    gaussDeriv = gaussianDerivKernel(sigma);
    flip_gaussDeriv =(double * ) malloc(w * sizeof(double));

//#pragma omp parallel for{
    for(i = w-1, j = 0; i >= 0; i--, j++){

        flip_gaussDeriv[j] = gaussDeriv[i];
        //printf("%d \n", w);
    }

    /*
    image = (int*)malloc(sizeof(int)*height*width);
    int j;
    for (j = 0; j < height *width; j++){
        image[j] = j;
        printf("%d ", image[j]);


    }
    printf("\n");
*/
    int block_dim;
    if(atoi(argv[3]) == 8 || atoi(argv[3]) == 16 ||atoi(argv[3]) == 32)
        block_dim = atoi(argv[3]);

    else
        block_dim = 16;
    //double kernel[] = {0.166379, 0.667243, 0.166379  };
    //double dkernel[] = {-.5, -0 , .5};
    int k_width = w;
    
    int *temp_hor;
    int *temp_ver;
    int *d_image;
    int *d_temp_hor;
    int *d_temp_ver;
    double *d_kernel;
    double *d_dkernel;
    int *d_grad;
    int *d_mag;
    int *temp_grad;
    int* temp_mag;
    int* d_edge;
    int* temp_edge;
    //Malloc for Host 
    temp_hor = (int*)malloc(sizeof(int)*height*width);
    //horizontal_grad = (int*)malloc(sizeof(int)*height*width);
    temp_ver = (int*)malloc(sizeof(int)*height*width);
    temp_grad = (int*)malloc(sizeof(int)*height*width);
    temp_mag = (int*)malloc(sizeof(int)*height*width);
    temp_edge = (int*)malloc(sizeof(int)*height*width);


    // Malloc for DEVICE GPU
    hipMalloc((void **)&d_image,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_hor,sizeof(int)*width*height);
    //hipMalloc((void **)&d_horizontal_grad,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_ver,sizeof(int)*width*height);
    
    hipMalloc((void **)&d_grad,sizeof(int)*width*height);
    hipMalloc((void **)&d_mag,sizeof(int)*width*height);
    hipMalloc((void **)&d_edge,sizeof(int)*width*height);



    hipMalloc((void **)&d_kernel,sizeof(double)*k_width);
    hipMalloc((void **)&d_dkernel,sizeof(double)*k_width);

    //hipMalloc((void **)&d_deriv_kernel,sizeof(int)*k_width);

    gettimeofday(&start_memCpy, NULL);
    //copy the items from CPU to GPU
    hipMemcpy(d_image, image,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, gauss,sizeof(double)*k_width,hipMemcpyHostToDevice);
    hipMemcpy(d_dkernel, flip_gaussDeriv,sizeof(double)*k_width,hipMemcpyHostToDevice);
    gettimeofday(&end_memCpy, NULL);
    //hipMemcpy(d_deriv_kernel,deriv_kernel,sizeof(int)*k_width,hipMemcpyHostToDevice);


    //Horizontal
    //int block_dim=atof(argv[3]);
    //int block_dim = 32;
    dim3 dmBlock(block_dim, block_dim, 1);
    dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);

    //printf("Height: %d\n", height);
    //printf("Width: %d\n", width);


    //horizontal gradient
    gettimeofday(&start_hor, NULL);

    convolveVert<<<dmGrid,dmBlock>>>(d_image,d_kernel, k_width, height,width, d_temp_hor);	
    convolveHoriz<<<dmGrid,dmBlock>>>(d_temp_hor,d_dkernel, k_width, height,width, d_temp_hor);	
/*
    sharedConvol<<<dmGrid, dmBlock,sizeof(int)*block_dim*block_dim>>>(d_image,d_temp_hor, d_kernel, height, width, k_width, 1);//vertical
    sharedConvol<<<dmGrid, dmBlock,sizeof(int)*block_dim*block_dim>>>(d_temp_hor,d_temp_hor, d_dkernel, height, width, 1, k_width);//horizontal
*/

    hipDeviceSynchronize();

    gettimeofday(&end_hor, NULL);

    //vertical gradient
    gettimeofday(&start_ver, NULL);

    convolveHoriz<<<dmGrid,dmBlock>>>(d_image,d_kernel, k_width, height, width, d_temp_ver);	
    convolveVert<<<dmGrid,dmBlock>>>(d_temp_ver,d_dkernel, k_width, height,width, d_temp_ver);	

/*
    sharedConvol<<<dmGrid, dmBlock,sizeof(int)*block_dim*block_dim>>>(d_image,d_temp_ver, d_kernel, height, width, 1, k_width);//horizontal
    sharedConvol<<<dmGrid, dmBlock,sizeof(int)*block_dim*block_dim>>>(d_temp_ver,d_temp_ver, d_dkernel, height, width, k_width, 1);//vertical
 */   

    hipDeviceSynchronize();

    gettimeofday(&end_ver, NULL);




    //copy to be saved

    
    gettimeofday(&start_dir, NULL);
    direction<<<dmGrid, dmBlock>>>(d_temp_hor, d_temp_ver, d_grad, height, width);
    hipDeviceSynchronize();
    gettimeofday(&end_dir, NULL);

    gettimeofday(&start_mag, NULL);
    magnitude<<<dmGrid, dmBlock>>>(d_temp_hor, d_temp_ver, d_mag, height, width);
    hipDeviceSynchronize();
    gettimeofday(&end_mag, NULL);


    hipMemcpy(d_edge, temp_mag,sizeof(int)*width*height,hipMemcpyHostToDevice);

    gettimeofday(&start_supp, NULL);
    suppression<<<dmGrid, dmBlock>>>(d_mag, d_grad, height, width, d_edge);
    hipDeviceSynchronize();
    gettimeofday(&end_supp, NULL);

    gettimeofday(&start_sort, NULL);
    thrust::device_ptr<int> thr_d(d_edge);
	thrust::device_vector<int>d_sup_vec(thr_d,thr_d+(height*width));
    thrust::sort(d_sup_vec.begin(),d_sup_vec.end());
	int index = (int)(.9*height*width);
	int t_hi = d_sup_vec[index];
    int t_lo =t_hi/2;
    gettimeofday(&end_sort, NULL);
    
    gettimeofday(&start_hys, NULL);
    hysteresis<<<dmGrid, dmBlock>>>(d_edge, height, width, t_hi, t_lo);
    hipDeviceSynchronize();
    gettimeofday(&end_hys, NULL);

    gettimeofday(&start_edge, NULL);
    edgeLinking<<<dmGrid, dmBlock>>>(d_edge, height, width, d_edge );
    hipDeviceSynchronize();
    gettimeofday(&end_edge, NULL);

    gettimeofday(&start_memCpydevice, NULL);
    hipMemcpy(temp_hor, d_temp_hor,sizeof(int)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_ver, d_temp_ver,sizeof(int)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_grad, d_grad,sizeof(int)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_mag, d_mag,sizeof(int)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_edge, d_edge,sizeof(int)*width*height,hipMemcpyDeviceToHost);
    gettimeofday(&end_memCpydevice, NULL);



    
    printf("End of Func\n");
/*
    for(i = 0; i < height * width; i++){
       printf("%d ", temp_hor[i]);
    }
    printf("\n");
*/
    gettimeofday(&start_edge_write, NULL);
    write_image_template("ansHor", temp_hor, width, height);
    write_image_template("ansVer", temp_ver, width, height);
    write_image_template("ansMag", temp_mag, width, height);
    write_image_template("ansDir", temp_grad, width, height);
    write_image_template("ansEdg", temp_edge, width, height);
    gettimeofday(&end_edge_write, NULL);



    hipDeviceSynchronize();
    gettimeofday(&end_total, NULL);


    printf("Total time :%ld\n", ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec)));
    printf("Total - File-IO: %ld\n", ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec) - ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)) - ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec))));
    printf("File IO image read: %ld\n", ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)));
    printf("File IO image write: %ld\n", ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec)));
    printf("Host to Device: %ld\n", ((end_memCpy.tv_sec * 1000000 + end_memCpy.tv_usec) - (start_memCpy.tv_sec * 1000000 + start_memCpy.tv_usec)));
    printf("Horizontal Convolution :%ld\n", ((end_hor.tv_sec * 1000000 + end_hor.tv_usec) - (start_hor.tv_sec * 1000000 + start_hor.tv_usec)));
    printf("Vertical Convolution: %ld\n", ((end_ver.tv_sec * 1000000 + end_ver.tv_usec) - (start_ver.tv_sec * 1000000 + start_ver.tv_usec)));
    printf("Magnitude Kernel: %ld\n", ((end_mag.tv_sec * 1000000 + end_mag.tv_usec) - (start_mag.tv_sec * 1000000 + start_mag.tv_usec)));
    printf("Gradient Kernel: %ld\n", ((end_dir.tv_sec * 1000000 + end_dir.tv_usec) - (start_dir.tv_sec * 1000000 + start_dir.tv_usec)));
    printf("Thrust Sorting: %ld\n", ((end_sort.tv_sec * 1000000 + end_sort.tv_usec) - (start_sort.tv_sec * 1000000 + start_sort.tv_usec)));
    printf("Suppression Kernel: %ld\n", ((end_supp.tv_sec * 1000000 + end_supp.tv_usec) - (start_supp.tv_sec * 1000000 + start_supp.tv_usec)));
    printf("Hysteresis Kernel: %ld\n", ((end_hys.tv_sec * 1000000 + end_hys.tv_usec) - (start_hys.tv_sec * 1000000 + start_hys.tv_usec)));
    printf("Edge Linking Kernel: %ld\n", ((end_edge.tv_sec * 1000000 + end_edge.tv_usec) - (start_edge.tv_sec * 1000000 + start_edge.tv_usec)));
    printf("Device to Host: %ld\n", ((end_memCpydevice.tv_sec * 1000000 + end_memCpydevice.tv_usec) - (start_memCpydevice.tv_sec * 1000000 + start_memCpydevice.tv_usec)));


    free(temp_hor);
    free(temp_ver);
    hipFree(d_image);
    hipFree(d_temp_hor);
    hipFree(d_temp_ver);
    hipFree(d_kernel);
    hipFree(d_dkernel);
    hipFree(d_grad);
    hipFree(d_mag);
    free(temp_grad);
    free(temp_mag);
    hipFree(d_edge);
    free(temp_edge);


    return 0;




}