#include "hip/hip_runtime.h"
//This will be the c File for program 1
/*
Author: Robert Hughes
This program will be creaed by using the documentation that was given to us in class along with using the code from ecpe 124 that Dr.Pallipuram gave us last semester


*/
#include <stdio.h>
#include <stddef.h>
#include <time.h>
#include <math.h>

#include "Gaussian.h"
#include "image.h"
#include "GPUconvolution.h"
#include "testConvolve.h"
//#include "image_template.h" not for this assignment C cannot handle the templates
#include <hip/hip_runtime.h>
//#include "Gaussian.h"

//Function Definition


//Function to call to get the Gaussian Kernel 


//Can use sigma as a constant to run the code
//#define sigma .6
#define WINDOW 7

int main( int argc, char** argv ){

    int block_dim = 16;
    
    
//set up the omp environment
//printf ("The number of processors is: %d\n", num_procs);
//omp_set_num_threads(atof(argv[3]));
//printf ("The number of threads is: %f\n", atof(argv[3]));


//allow Sigma to be put in the command line to be set
clock_t start, end;
start = clock();
double cpu_time_used;
//char filename[] = "Corners.csv";

//FILE *fp;
    int *temp_hor;
    int *temp_ver;
    int *d_image;
    int *d_temp_hor;
    int *d_temp_ver;
    double *d_kernel;
    int *d_vert;
    int *d_horiz;
    int *d_grad;
    int *d_mag;

    double *flip_gaussDeriv;    
    double *d_derivKernel;






float sigma = atof(argv[2]);

//integers for the loops
int i =0;
int j =0;

//get the size of the kernel so we can store it after the functions are preformed
int w = 2*(round(2.5*sigma -.5))+1;
//printf("w= %d \n", w);
//allocate these according to the width of the kernel
/*
used these to test the convolution because they are easier to check the work to make sure that the function is behaving correctly
double temp[] = {0, 1, 2, 3, 
                4, 5, 6, 7,
                8, 9, 10, 11, 
                12, 13, 14, 15};

double kernel[] = {.25, .5, .25};
*/
//pre-define these so that we can pull the image in using the library provided to us
int height;
int width;
int *image;

//calculate the two kernels we need to run the program
double *gauss;// = (double*)malloc(sizeof(double)*w);
double *gaussDeriv;// = (double*)malloc(sizeof(double)*w);


gauss = gaussianKernel(sigma);

gaussDeriv = gaussianDerivKernel(sigma);

//printf("Hello\n");

read_image_template(argv[1], &image, &width, &height);


/*
printf("%d\n", height);
printf("%d\n", width);
for(int i = 0; i < height*width; i++){
    printf("%d\n", image[i]);
}
*/
//w=3;


//free( array );
//free( otherarray );

//Flip the Gaussian Derivative Kernel since it is not symetric
flip_gaussDeriv =(double * ) malloc(w * sizeof(double));

//#pragma omp parallel for{
    for(i = w-1, j = 0; i >= 0; i--, j++){

    flip_gaussDeriv[j] = gaussDeriv[i];
        //printf("%d \n", w);
    }
//}
printf("Flipped Gaussian Derivative: ");

for(j = 0; j < w; j++){

        printf("%f ", flip_gaussDeriv[j]);

    }
    printf("\nGauss: ");
    for(j = 0; j < w; j++){
    
        printf("%f ", gauss[j]);

    }
    printf("\n");
  
free(gaussDeriv);//we can now free the gaussian derivative kernel since we have the new flipped kernel


dim3 dmBlock(block_dim, block_dim, 1);
dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);


//*****Section to compute the Grandient and Magnitude********

//preform the convolutions
    // Malloc for DEVICE GPU
    hipMalloc((void **)&d_image,sizeof(int)*width*height);
    hipMalloc((void **)&d_kernel,sizeof(double)*w);
    hipMalloc((void **)&d_derivKernel,sizeof(double)*w);

    hipMalloc((void **)&d_temp_hor,sizeof(int)*width*height);
    hipMalloc((void **)&d_horiz,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_ver,sizeof(int)*width*height);
    hipMalloc((void **)&d_vert,sizeof(int)*width*height);

    hipMalloc((void **)&d_kernel,sizeof(double)*w);
\

    hipMalloc((void **)&d_grad,sizeof(int)*width*height);
    hipMalloc((void **)&d_mag,sizeof(int)*width*height);

    //copy the items from CPU to GPU
    hipMemcpy(d_image, image,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, gauss,sizeof(int)*w,hipMemcpyHostToDevice);
    hipMemcpy(d_derivKernel, flip_gaussDeriv,sizeof(int)*w,hipMemcpyHostToDevice);


//compute the horizontal and vertical gradients
//make space for the arrays
int *Vertical;
int *Horizontal;

Vertical = (int*)malloc(height*width * sizeof(int));
Horizontal = (int*)malloc(height*width * sizeof(int));
//tempV = convolveVert( image, gauss, w, height, width );

// create characer arrays to name the files for when we use the image write
char tempVer[] = "temp_vertical";
char tempHor[] = "temp_horizontal";
char ver[] = "vertical";
char hor[] = "horizontal";
char grad[] = "gradient";
char mag[] = "magnitude";

//Horiz
convolveVert<<<dmGrid,dmBlock>>>(d_image, d_kernel, w, height,width, d_horiz);	
//hipMemcpy(Horizontal, d_temp_hor,sizeof(int)*height*width,hipMemcpyDeviceToHost);

convolveHoriz<<<dmGrid,dmBlock>>>(d_horiz,d_derivKernel, w, height,width, d_horiz);	
hipMemcpy(Horizontal, d_horiz,sizeof(int)*height*width,hipMemcpyDeviceToHost);
for(i = 0; i < height * width; i++){
    printf("%d ", Horizontal[i]);
 }
write_image_template(hor, Horizontal, width, height);


/*

//Vertical
convolveVert<<<dmGrid,dmBlock>>>(d_image,d_kernel, w, height, width,d_temp_ver);
hipMemcpy(Vertical, d_temp_hor,sizeof(int)*height*width,hipMemcpyDeviceToHost);

convolveVert<<<dmGrid,dmBlock>>>(d_temp_hor, d_derivKernel, w, height,width, d_vert); 
hipMemcpy(Vertical, d_vert,sizeof(int)*height*width,hipMemcpyDeviceToHost);

write_image_template(ver, Vertical, width, height);
*/

/*

//****Calculate the Maginitude and Gradient****
//allocate the memory
int * Magnitude = malloc(height*width * sizeof(int));
int * Gradient = malloc(height*width * sizeof(int));

//can put these two together in one loop to try to increase the speed, could preform loop unrolling however it is trivial and we are going to use openMP in the future anyway
for(i = 0; i < height*width; i++){
    Gradient[i] = atan2(Horizontal[i], Vertical[i]);
    //using the euclidean distance for the magnitude
    Magnitude[i] = sqrt((Vertical[i]*Vertical[i])+(Horizontal[i]*Horizontal[i]));

}

//write out the files

write_image_template(mag, Magnitude, width, height);
write_image_template(grad, Gradient, width, height);


//printf("Done first part\n");

//preform the non-max suppression
char sup[] = "Suppression";


int * edges = malloc(height*width * sizeof(int));
edges = suppression(Magnitude, Gradient, height, width);
//printf("Finished the supression\n");

write_image_template(sup, edges, width, height);


//preform the hysteresis
char hys[] = "Hyteresis";

edges = hysteresis(edges, height, width);
//printf("Finished the hyteresis\n");

write_image_template(hys, edges, width, height);


//preform the edge linking
char edg[] = "Edges";

edges = edgeLinking(edges, height, width);
//printf("Finished the edge linking\n");

write_image_template(edg, edges, width, height);

int * corners = malloc(height*width * sizeof(int));
corners = cornersList(Vertical, Horizontal, height, width, WINDOW);
write_image_template("corners", corners, width, height);


/*
fp = fopen(filename, "w+");
fprintf(fp, "Corners");
for(i = 0; i < atof(argv[3]); i ++){
    fprintf(fp, "\n%d", i+1);
}
*/
/*debugger code
for(int i = 0; i < height*width; i++){
    printf("%f", Magnitude[i]);
}
*/
//printf("finished!\n");
free(Vertical);
free(Horizontal);
free(flip_gaussDeriv);
free(gauss);
/*
free(Magnitude);
free(Gradient);
free (corners);
free(edges);
*/


//printf("Program Executed: Egde found.\n");
end = clock();
cpu_time_used = ((double)(end - start))/ CLOCKS_PER_SEC;
//printf("Time taken to execute: %f\n", cpu_time_used);
//printf("%s, %f, %f \n", argv[3], sigma, cpu_time_used);


    return 0;
}
