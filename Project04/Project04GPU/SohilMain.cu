#include "hip/hip_runtime.h"
/************************************************
*	   					*
*		Author: Sohil Singh		*
*		Project 2: Corner Detection	* 
*						*
*		Date: 2/06/2019			*
*						*
************************************************/

#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <errno.h>
#include <math.h>
#include <ctype.h>
#include "image_template.h"
#include "gaussian.h"
#include "gaussian_derivative.h"
#include "convolution_horizontal.h"
#include "convolution_horizontal_shared.h"
#include "convolution_vertical.h"
#include "convolution_vertical_shared.h"
#include "magnitude.h"
#include "direction.h"
#include "suppress.h"
#include "comparator.h"
#include "hysteresis.h"
#include "edges.h"
//#include "corners.h"
//#include "output_corners.h"
#include <time.h>
#include <sys/time.h>

int main(int argc, char *argv[])
{
	
	// Time each kernel
	// Communication times memcpy, device to host, host to device
	// No file i/o

	// Pointer logic
	// int a = 7; stores value to memory address as 7
	// int *a_ptr = &a; get memory address where a value is located
	// int a_address = a_ptr; a_address gets memory address a_ptr stores
	// int a_value = *a_ptr; get value at a_ptr memory address
	// To change a variable in a function must call by reference by passing the 		// address of the variable
	// Read input image file

	hipFree(0);
	// CUDA Device synchronize
	//	
	int j,t,comp_time,ete_time;
	int height;
	int width, size,ninety_percentile_index;
	char sys_arr[2];
	// create 2D array to hold input image	
	float *input_image=NULL,t_hi,t_lo;
	
	float sigma=atof(argv[2]);
	int block_size = atoi(argv[3]);
	int gaussian_width,gd_width;
	float *hor_gaussian=NULL,*hor_gaussian_derivative=NULL,*temp_horizontal=NULL,*horizontal=NULL,*temp_vertical=NULL,*vertical=NULL,*mag_output=NULL,*dir_output=NULL,*sup_output=NULL,*sort_sup_output=NULL,*hyst_output=NULL,*edges_output=NULL,*corner_output=NULL,*output_image=NULL;
	float *d_input_image, *d_hor_gaussian, *d_hor_gaussian_deriv, *d_temp_horizontal, *d_horizontal; 
	//"/Users/sss/hpc_ecpe_251/Lenna_org_256.pgm"
	float *d_temp_vertical, *d_vertical;
	float *d_mag_output, *d_dir_output;
	float *d_sup_output, *d_hyst_output,*d_edges_output;
	char *input_filepath = argv[1];
	// printf("Filepath: %s\n", input_filepath);		// Test line
	//char *output_filepath = "/Users/sss/hpc_ecpe_251/output.pgm";
	// printf("Filepath: %s\n", output_filepath);		// Test line
	//omp_set_num_threads(atoi(argv[3])); //Set num of threads eq. to procs
	
	int iterations = atoi(argv[1]);
	// ete = end to end
	// comp = computation
  	struct timeval start_ete, end_ete, start_comp, end_comp, start_fileio1, end_fileio1;
	struct timeval start_fileio2, end_fileio2, start_h2d, end_h2d, start_conv_hor, end_conv_hor;
	struct timeval start_conv_ver, end_conv_ver, start_mag, end_mag, start_dir, end_dir, start_sort, end_sort;
	struct timeval start_sup, end_sup, start_edge, end_edge, start_hyst, end_hyst, start_d2h, end_d2h;

  	//gettimeofday(&start_ete, NULL);

	// Read in image
        gettimeofday(&start_fileio1, NULL);
        read_image_template(input_filepath,&input_image,&width,&height);
        gettimeofday(&end_fileio1, NULL);
	
//	malloc GPU buffers
	hipMalloc((void **)&d_input_image, sizeof(float)*height*width);
	hipMalloc((void **)&d_hor_gaussian, sizeof(float)*gaussian_width);
	hipMalloc((void **)&d_hor_gaussian_deriv, sizeof(float)*gd_width);
	hipMalloc((void **)&d_temp_horizontal, sizeof(float)*height*width);
	hipMalloc((void **)&d_horizontal, sizeof(float)*height*width);
	hipMalloc((void **)&d_temp_vertical, sizeof(float)*height*width);
        hipMalloc((void **)&d_vertical, sizeof(float)*height*width);	
	hipMalloc((void **)&d_mag_output, sizeof(float)*height*width);
        hipMalloc((void **)&d_dir_output, sizeof(float)*height*width);
	hipMalloc((void **)&d_sup_output, sizeof(float)*height*width);
	hipMalloc((void **)&d_hyst_output, sizeof(float)*height*width);
	hipMalloc((void **)&d_edges_output, sizeof(float)*height*width);	

       //malloc host buffers
	horizontal=(float*)malloc(sizeof(float)*(height*width));	
	temp_horizontal=(float*)malloc(sizeof(float)*(height*width));
	temp_vertical=(float*)malloc(sizeof(float)*(height*width));
	vertical=(float*)malloc(sizeof(float)*(height*width));	
	mag_output=(float*)malloc(sizeof(float)*(height*width));
        dir_output=(float*)malloc(sizeof(float)*(height*width));
	sup_output=(float*)malloc(sizeof(float)*(height*width));	
	hyst_output=(float*)malloc(sizeof(float)*(height*width));
	edges_output=(float*)malloc(sizeof(float)*(height*width));	

	gettimeofday(&start_ete, NULL);

	
        // Generate horizontal gaussian kernel
        gaussian(sigma,&hor_gaussian,&gaussian_width);

        // Generate horizontal gaussian derivative kernel
        gaussian_derivative(sigma,&hor_gaussian_derivative,&gd_width);

//	Transfer gaussian data to GPGPU device
	gettimeofday(&start_h2d, NULL);
	hipMemcpy(d_input_image, input_image, sizeof(float)*height*width, hipMemcpyHostToDevice);
	hipMemcpy(d_hor_gaussian, hor_gaussian, sizeof(float)*gaussian_width, hipMemcpyHostToDevice);
	hipMemcpy(d_hor_gaussian_deriv, hor_gaussian_derivative, sizeof(float)*gd_width, hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	gettimeofday(&end_h2d, NULL);
	// write_image_template("Input.pgm",input_image,width,height);
	//printf("Input[0][0] = %f",input_image[65535]);
	//printf("Input[0][5] = %d",0*0);
	
	dim3 dimBlock(block_size, block_size);
	dim3 dimGrid(height/block_size, width/block_size);
	
	// Convolution horizontal
	gettimeofday(&start_conv_hor, NULL);
//	convolution_vertical<<<dimGrid, dimBlock>>>(d_input_image,d_temp_horizontal,d_hor_gaussian,gd_width,1,height,width);
//	convolution_horizontal<<<dimGrid, dimBlock>>>(d_temp_horizontal,d_horizontal,d_hor_gaussian_deriv,gaussian_width,height,width);
	convolution_vertical_shared<<<dimGrid, dimBlock,sizeof(float)*block_size*block_size>>>(d_input_image,d_temp_horizontal,d_hor_gaussian,gd_width,1,height,width);
	convolution_horizontal_shared<<<dimGrid, dimBlock,sizeof(float)*block_size*block_size>>>(d_temp_horizontal,d_horizontal,d_hor_gaussian_deriv,1,gaussian_width,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_conv_hor, NULL);

	// Convolution vertical
	gettimeofday(&start_conv_ver, NULL);
//	convolution_horizontal<<<dimGrid, dimBlock>>>(d_input_image,d_temp_vertical,d_hor_gaussian,gaussian_width,height,width);
//	convolution_vertical<<<dimGrid, dimBlock>>>(d_temp_vertical,d_vertical,d_hor_gaussian_deriv,gd_width,1,height,width);
	convolution_horizontal_shared<<<dimGrid, dimBlock,sizeof(float)*block_size*block_size>>>(d_input_image,d_temp_vertical,d_hor_gaussian,1,gaussian_width,height,width);
	convolution_vertical_shared<<<dimGrid, dimBlock,sizeof(float)*block_size*block_size>>>(d_temp_vertical,d_vertical,d_hor_gaussian_deriv,gd_width,1,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_conv_ver, NULL);	

	// Magnitude
	gettimeofday(&start_mag, NULL);	
	magnitude<<<dimGrid, dimBlock>>>(d_mag_output,d_horizontal,d_vertical,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_mag, NULL);

	// Direction
	gettimeofday(&start_dir, NULL);
	direction<<<dimGrid, dimBlock>>>(d_dir_output,d_horizontal,d_vertical,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_dir, NULL);

	// Non-max suppression
	//memcpy(sup_output, mag_output, height*width*sizeof(float));
	gettimeofday(&start_sup, NULL);
//	hipMemcpy(d_sup_output, d_mag_output, sizeof(float)*height*width, hipMemcpyDeviceToDevice);
	suppress<<<dimGrid, dimBlock>>>(d_sup_output,d_mag_output,d_dir_output,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_sup, NULL);
	
//	hipDeviceSynchronize();	
	gettimeofday(&start_sort, NULL);
	thrust::device_ptr<float> thr_d(d_sup_output);
	thrust::device_vector<float>d_sup_vec(thr_d,thr_d+(height*width));
	thrust::sort(d_sup_vec.begin(),d_sup_vec.end());
	int index = (int)(.9*height*width);
	float th_high = d_sup_vec[index];
	float th_low =th_high/5;
//	cudaDeviceSychronize();
	gettimeofday(&end_sort, NULL);

	// Hysteresis
	// Sort values
	
	gettimeofday(&start_hyst, NULL);
	hysteresis<<<dimGrid, dimBlock>>>(d_hyst_output,d_sup_output,height,width,th_high,th_low);
	hipDeviceSynchronize();
	gettimeofday(&end_hyst, NULL);

	// Edges
	gettimeofday(&start_edge, NULL);
	edges<<<dimGrid, dimBlock>>>(d_edges_output,d_hyst_output,height,width);
	hipDeviceSynchronize();
	gettimeofday(&end_edge, NULL);
	
	// Device to host
	gettimeofday(&start_d2h, NULL);
	hipMemcpy(temp_horizontal, d_temp_horizontal, sizeof(float)*height*width, hipMemcpyDeviceToHost);
        hipMemcpy(horizontal, d_horizontal, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(temp_vertical, d_temp_vertical, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(vertical, d_vertical, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(mag_output, d_mag_output, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(dir_output, d_dir_output, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(sup_output, d_sup_output, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipMemcpy(hyst_output, d_hyst_output, sizeof(float)*height*width, hipMemcpyDeviceToHost);	
	hipMemcpy(edges_output, d_edges_output, sizeof(float)*height*width, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	gettimeofday(&end_d2h, NULL);
       
	// Write out images; file output 2
	gettimeofday(&start_fileio2, NULL);
        write_image_template("Temp_Horizontal.pgm",temp_horizontal,width,height);
        write_image_template("Horizontal.pgm",horizontal,width,height);
	write_image_template("Temp_Vertical.pgm",temp_vertical,width,height);
	write_image_template("Vertical.pgm",vertical,width,height);
	write_image_template("Magnitude.pgm",mag_output,width,height);
	write_image_template("Gradient.pgm",dir_output,width,height);
	write_image_template("Suppress.pgm",sup_output,width,height);
	write_image_template("Hysteresis.pgm",hyst_output,width,height);
	write_image_template("Edges.pgm",edges_output,width,height);
//	hipDeviceSynchronize();
	gettimeofday(&end_fileio2, NULL);


/*
//	corners(&corner_output,vertical,horizontal,height,width,7);
	
//	output_corners(&output_image, corner_output, input_image, height, width);

*/
//	gettimeofday(&end_comp, NULL);
//	write_image_template("Corner.pgm",output_image,width,height);
	hipDeviceSynchronize();
	gettimeofday(&end_ete, NULL);
	/*
	start_fileio1, end_fileio1;
        struct timeval start_fileio2, end_fileio2, start_h2d, end_h2d, start_conv_hor, end_conv_hor;
        struct timeval start_conv_ver, end_conv_ver, start_mag, end_mag, start_dir, end_dir, start_sort, end_sort;
        struct timeval start_sup, end_sup, start_edge, end_edge, start_hyst, end_hyst, start_d2h, end_d2h;
	*/
//	printf("%ld,", ((end_comp.tv_sec * 1000000 + end_comp.tv_usec) - (start_comp.tv_sec * 1000000 + start_comp.tv_usec)));
	printf("%ld\n", ((end_ete.tv_sec * 1000000 + end_ete.tv_usec) - (start_ete.tv_sec * 1000000 + start_ete.tv_usec)));
	printf("%ld\n", ((end_ete.tv_sec * 1000000 + end_ete.tv_usec) - (start_ete.tv_sec * 1000000 + start_ete.tv_usec) - ((end_fileio1.tv_sec * 1000000 + end_fileio1.tv_usec) - (start_fileio1.tv_sec * 1000000 + start_fileio1.tv_usec)) - ((end_fileio2.tv_sec * 1000000 + end_fileio2.tv_usec) - (start_fileio2.tv_sec * 1000000 + start_fileio2.tv_usec))));
	printf("%ld\n", ((end_fileio1.tv_sec * 1000000 + end_fileio1.tv_usec) - (start_fileio1.tv_sec * 1000000 + start_fileio1.tv_usec)));
	printf("%ld\n", ((end_fileio2.tv_sec * 1000000 + end_fileio2.tv_usec) - (start_fileio2.tv_sec * 1000000 + start_fileio2.tv_usec)));
	printf("%ld\n", ((end_h2d.tv_sec * 1000000 + end_h2d.tv_usec) - (start_h2d.tv_sec * 1000000 + start_h2d.tv_usec)));
        printf("%ld\n", ((end_fileio2.tv_sec * 1000000 + end_fileio2.tv_usec) - (start_fileio2.tv_sec * 1000000 + start_fileio2.tv_usec)));
	printf("%ld\n", ((end_conv_hor.tv_sec * 1000000 + end_conv_hor.tv_usec) - (start_conv_hor.tv_sec * 1000000 + start_conv_hor.tv_usec)));
        printf("%ld\n", ((end_conv_ver.tv_sec * 1000000 + end_conv_ver.tv_usec) - (start_conv_ver.tv_sec * 1000000 + start_conv_ver.tv_usec)));
        printf("%ld\n", ((end_mag.tv_sec * 1000000 + end_mag.tv_usec) - (start_mag.tv_sec * 1000000 + start_mag.tv_usec)));
        printf("%ld\n", ((end_dir.tv_sec * 1000000 + end_dir.tv_usec) - (start_dir.tv_sec * 1000000 + start_dir.tv_usec)));
	printf("%ld\n", ((end_sort.tv_sec * 1000000 + end_sort.tv_usec) - (start_sort.tv_sec * 1000000 + start_sort.tv_usec)));
        printf("%ld\n", ((end_sup.tv_sec * 1000000 + end_sup.tv_usec) - (start_sup.tv_sec * 1000000 + start_sup.tv_usec)));
	printf("%ld\n", ((end_hyst.tv_sec * 1000000 + end_hyst.tv_usec) - (start_hyst.tv_sec * 1000000 + start_hyst.tv_usec)));
        printf("%ld\n", ((end_edge.tv_sec * 1000000 + end_edge.tv_usec) - (start_edge.tv_sec * 1000000 + start_edge.tv_usec)));
	
	return 0;


}


