#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stddef.h>
#include <time.h>
#include <omp.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "image.h"

#include "testConvolve.h"
#include "GPUMagDir.h"
//#include "image_template.h" not for this assignment C cannot handle the templates



//Can use sigma as a constant to run the code
//#define sigma .6

int main(int argc, char **argv)
{

    int height;
    int width;
    int *vert;
    int *horiz;
    int *mag;
    int *grad;

    //read_image_template(argv[1], &vert, &width, &height);
    //read_image_template(argv[2], &horiz, &width, &height);


    


    int block_dim = 16;

    double kernel[] = {.25, .5, .25 };
    int k_width = 3;
    
    int *temp_hor;
    int *temp_ver;
    int *d_image;
    int *d_temp_hor;
    int *d_temp_ver;
    double *d_kernel;
    int *d_vert;
    int *d_horiz;
    int *d_grad;
    int *d_mag;
    
    //Malloc for Host 
    temp_hor = (int*)malloc(sizeof(int)*height*width);
    //horizontal_grad = (int*)malloc(sizeof(int)*height*width);
    temp_ver = (int*)malloc(sizeof(int)*height*width);
    //vertical_grad = (int*)malloc(sizeof(int)*height*width);

    mag = (int*)malloc(sizeof(int)*height*width);
    grad = (int*)malloc(sizeof(int)*height*width);





    // Malloc for DEVICE GPU
    hipMalloc((void **)&d_image,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_hor,sizeof(int)*width*height);
    //hipMalloc((void **)&d_horizontal_grad,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_ver,sizeof(int)*width*height);
    //hipMalloc((void **)&d_vertical_grad,sizeof(int)*width*height);

    hipMalloc((void **)&d_kernel,sizeof(double)*k_width);
    //hipMalloc((void **)&d_deriv_kernel,sizeof(int)*k_width);


    hipMalloc((void **)&d_vert,sizeof(int)*width*height);
    hipMalloc((void **)&d_horiz,sizeof(int)*width*height);
    hipMalloc((void **)&d_grad,sizeof(int)*width*height);
    hipMalloc((void **)&d_mag,sizeof(int)*width*height);

    //copy the items from CPU to GPU
    hipMemcpy(d_vert, vert,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(d_horiz, horiz,sizeof(int)*width*height,hipMemcpyHostToDevice);
    //hipMemcpy(d_deriv_kernel,deriv_kernel,sizeof(int)*k_width,hipMemcpyHostToDevice);


    //Horizontal
    //int block_dim=atof(argv[3]);
    //int block_dim = 32;
    dim3 dmBlock(block_dim, block_dim, 1);
    dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);

    //printf("Height: %d\n", height);
    //printf("Width: %d\n", width);


    magnitude<<<dmGrid,dmBlock>>>(d_horiz, d_vert, d_mag, height, width);	
    hipMemcpy(mag, d_mag,sizeof(int)*width*height,hipMemcpyDeviceToHost);

    direction<<<dmGrid,dmBlock>>>(d_horiz, d_vert, d_grad, height, width);	
    hipMemcpy(grad, d_grad,sizeof(int)*width*height,hipMemcpyDeviceToHost);

    //convoultion<<<dmGrid,dmBlock>>>(d_temp_hor,d_horizontal_grad,d_deriv_kernel,height,width,1,k_width); 

    printf("End of Func");
    int i;
    for(i = 0; i < height * width; i++){
       // printf("%d ", grad[i]);
    }
    printf("\n\n\n\n");
    for(i = 0; i < height * width; i++){
        //printf("%d ", mag[i]);
    }
    write_image_template("mag", mag, width, height);
    write_image_template("grad",grad, width, height);


    hipDeviceSynchronize();

    return 0;




}

