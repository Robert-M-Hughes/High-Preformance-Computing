/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
 #include <stdio.h>
 #include <stdlib.h>
 #include <hip/hip_runtime.h>
 #include <hiprand/hiprand.h>
 #include <time.h>
 #include <sys/time.h>
 
 #define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__);\
     return EXIT_FAILURE;}} while(0)
 #define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
     printf("Error at %s:%d\n",__FILE__,__LINE__);\
     return EXIT_FAILURE;}} while(0)
 
 int main(int argc, char *argv[])
 {

    struct timeval start_h2d, end_h2d, start_d2h, end_d2h;


     int n;
     n = 1000 * 256 * atof(argv[1]); //256 * 1000 floats per mB
     size_t i;
     hiprandGenerator_t gen;
     float *devData, *hostData;
 
     /* Allocate n floats on host */
     hostData = (float *)calloc(n, sizeof(float));
 
     /* Allocate n floats on device */
     CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));
 
     /* Create pseudo-random number generator */
     CURAND_CALL(hiprandCreateGenerator(&gen, 
                 HIPRAND_RNG_PSEUDO_DEFAULT));
     
     /* Set seed */
     CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                 1234ULL));
 
     /* Generate n floats on device */
     CURAND_CALL(hiprandGenerateUniform(gen, devData, n));
 
     /* Copy device memory to host */
     CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
         hipMemcpyDeviceToHost));
 
     /* Show result 
     for(i = 0; i < n; i++) {
         printf("%1.4f ", hostData[i]);
     }
     printf("\n");
    */

    float* rando = (float*)malloc(n*sizeof(float));
    float* d_rand;
    hipMalloc((void **)&d_rand,sizeof(float)*n);


    gettimeofday(&start_h2d, NULL);
        hipMemcpy(d_rand, rando,sizeof(float)*n,hipMemcpyHostToDevice);
    gettimeofday(&end_h2d, NULL);


    gettimeofday(&start_d2h, NULL);
        hipMemcpy(rando, d_rand,sizeof(float)*n,hipMemcpyDeviceToHost);
    gettimeofday(&end_d2h, NULL);

    printf("%ld, %ld\n", ((end_h2d.tv_sec * 1000000 + end_h2d.tv_usec) - (start_h2d.tv_sec * 1000000 + start_h2d.tv_usec)),((end_d2h.tv_sec * 1000000 + end_d2h.tv_usec) - (start_d2h.tv_sec * 1000000 + start_d2h.tv_usec)));
    
     /* Cleanup */
     CURAND_CALL(hiprandDestroyGenerator(gen));
     CUDA_CALL(hipFree(devData));
     free(hostData);    
     return EXIT_SUCCESS;
 }