#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stddef.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/copy.h>

#include "image_template.h"
#include "Gaussian.h"
#include "GPUconvolution.h"
#include "GPUMagDir.h"
#include "GPUNonMaxSuppression.h"
#include "GPUHysteresis.h"
#include "GPUEdgeLinking.h"
#include "GPUCorners.h"
//#include "image_template.h" not for this assignment C cannot handle the templates


#define WINDOW 7

//Can use sigma as a constant to run the code
//#define sigma .6

int main(int argc, char **argv)
{




    struct timeval start_total, end_total, start_image_read, end_image_read, start_edge_write, 
    end_edge_write, start_memCpy, end_memCpy, start_hor, end_hor, start_ver, end_ver, start_mag,
    end_mag, start_dir, end_dir, start_sort, end_sort, start_supp, end_supp, start_edge, end_edge,
    start_hys, end_hys, start_memCpydevice, end_memCpydevice;

  	//gettimeofday(&start_ete, NULL);

    int i, j;
    int height;
    int width;
    float *image;

    float sigma = atof(argv[2]);
    int w = 2*(round(2.5*sigma -.5))+1;

    gettimeofday(&start_image_read, NULL);
        read_image_template(argv[1], &image, &width, &height);
    gettimeofday(&end_image_read, NULL);

	gettimeofday(&start_total, NULL);

    double *gauss;// = (double*)malloc(sizeof(double)*w);
    double *gaussDeriv;// = (double*)malloc(sizeof(double)*w);
    double *flip_gaussDeriv;

    gauss = gaussianKernel(sigma);
    gaussDeriv = gaussianDerivKernel(sigma);
    flip_gaussDeriv =(double * ) malloc(w * sizeof(double));

//#pragma omp parallel for{
    for(i = w-1, j = 0; i >= 0; i--, j++){

        flip_gaussDeriv[j] = gaussDeriv[i];
        //printf("%d \n", w);
    }

    /*
    image = (int*)malloc(sizeof(int)*height*width);
    int j;
    for (j = 0; j < height *width; j++){
        image[j] = j;
        printf("%d ", image[j]);


    }
    printf("\n");
*/
    int block_dim;
    /*
    if(atoi(argv[3]) == 8 || atoi(argv[3]) == 16 ||atoi(argv[3]) == 32)
        block_dim = atoi(argv[3]);

    else
        block_dim = 16;
        */
        block_dim = atoi(argv[3]);
    //double kernel[] = {0.166379, 0.667243, 0.166379  };
    //double dkernel[] = {-.5, -0 , .5};
    int k_width = w;
    
    float *temp_hor;
    float *temp_ver;
    float *d_image;
    float *d_temp_hor;
    float *d_temp_ver;
    double *d_kernel;
    double *d_dkernel;
    float *d_grad;
    float *d_mag;
    float *temp_grad;
    float* temp_mag;
    float* d_edge;
    float* temp_edge;
    float* d_corn;
    float* temp_corn;
    float* temp_indices;
    float* d_indices;
    //Malloc for Host 
    temp_hor = (float*)malloc(sizeof(float)*height*width);
    //horizontal_grad = (int*)malloc(sizeof(int)*height*width);
    temp_ver = (float*)malloc(sizeof(float)*height*width);
    temp_grad = (float*)malloc(sizeof(float)*height*width);
    temp_mag = (float*)malloc(sizeof(float)*height*width);
    temp_edge = (float*)malloc(sizeof(float)*height*width);
    temp_corn = (float*)malloc(sizeof(float)*height*width);
    temp_indices = (float*)malloc(sizeof(float)*height*width);


    // Malloc for DEVICE GPU
    hipMalloc((void **)&d_image,sizeof(int)*width*height);
    hipMalloc((void **)&d_temp_hor,sizeof(float)*width*height);
    //hipMalloc((void **)&d_horizontal_grad,sizeof(float)*width*height);
    hipMalloc((void **)&d_temp_ver,sizeof(float)*width*height);
    
    hipMalloc((void **)&d_grad,sizeof(float)*width*height);
    hipMalloc((void **)&d_mag,sizeof(float)*width*height);
    hipMalloc((void **)&d_edge,sizeof(float)*width*height);
    hipMalloc((void **)&d_corn,sizeof(float)*width*height);
    hipMalloc((void **)&d_indices,sizeof(float)*width*height);




    hipMalloc((void **)&d_kernel,sizeof(double)*k_width);
    hipMalloc((void **)&d_dkernel,sizeof(double)*k_width);

    //hipMalloc((void **)&d_deriv_kernel,sizeof(float)*k_width);

    gettimeofday(&start_memCpy, NULL);
    //copy the items from CPU to GPU
    hipMemcpy(d_image, image,sizeof(int)*width*height,hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, gauss,sizeof(double)*k_width,hipMemcpyHostToDevice);
    hipMemcpy(d_dkernel, flip_gaussDeriv,sizeof(double)*k_width,hipMemcpyHostToDevice);
    gettimeofday(&end_memCpy, NULL);
    //hipMemcpy(d_deriv_kernel,deriv_kernel,sizeof(int)*k_width,hipMemcpyHostToDevice);


    //Horizontal
    //int block_dim=atof(argv[3]);
    //int block_dim = 32;
    dim3 dmBlock(block_dim, block_dim, 1);
    dim3 dmGrid(ceil(height/block_dim), ceil(width/block_dim), 1);

    //printf("Height: %d\n", height);
    //printf("Width: %d\n", width);


    //horizontal gradient
    gettimeofday(&start_hor, NULL);
/*
    convolveVert<<<dmGrid,dmBlock>>>(d_image,d_kernel, k_width, height,width, d_temp_hor);	
    convolveHoriz<<<dmGrid,dmBlock>>>(d_temp_hor,d_dkernel, k_width, height,width, d_temp_hor);	
*/
    sharedConvol<<<dmGrid, dmBlock,sizeof(float)*block_dim*block_dim>>>(d_image,d_temp_hor, d_kernel, height, width, k_width, 1);//vertical
    sharedConvol<<<dmGrid, dmBlock,sizeof(float)*block_dim*block_dim>>>(d_temp_hor,d_temp_hor, d_dkernel, height, width, 1, k_width);//horizontal


    hipDeviceSynchronize();

    gettimeofday(&end_hor, NULL);

    //vertical gradient
    gettimeofday(&start_ver, NULL);

/*
    convolveHoriz<<<dmGrid,dmBlock>>>(d_image,d_kernel, k_width, height, width, d_temp_ver);	
    convolveVert<<<dmGrid,dmBlock>>>(d_temp_ver,d_dkernel, k_width, height,width, d_temp_ver);	
*/


    sharedConvol<<<dmGrid, dmBlock,sizeof(float)*block_dim*block_dim>>>(d_image,d_temp_ver, d_kernel, height, width, 1, k_width);//horizontal
    sharedConvol<<<dmGrid, dmBlock,sizeof(float)*block_dim*block_dim>>>(d_temp_ver,d_temp_ver, d_dkernel, height, width, k_width, 1);//vertical
    

    hipDeviceSynchronize();

    gettimeofday(&end_ver, NULL);




    //copy to be saved

    
    gettimeofday(&start_dir, NULL);
    direction<<<dmGrid, dmBlock>>>(d_temp_hor, d_temp_ver, d_grad, height, width);
    hipDeviceSynchronize();
    gettimeofday(&end_dir, NULL);

    gettimeofday(&start_mag, NULL);
    magnitude<<<dmGrid, dmBlock>>>(d_temp_hor, d_temp_ver, d_mag, height, width);
    hipDeviceSynchronize();
    gettimeofday(&end_mag, NULL);

    hipMemcpy(temp_mag, d_mag,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(d_edge, temp_mag,sizeof(float)*width*height,hipMemcpyHostToDevice);
    
    
  

    gettimeofday(&start_supp, NULL);
    suppression<<<dmGrid, dmBlock>>>(d_mag, d_grad, height, width, d_edge);
    hipDeviceSynchronize();
    gettimeofday(&end_supp, NULL);

    hipMemcpy(temp_edge, d_edge,sizeof(float)*width*height,hipMemcpyDeviceToHost);


    gettimeofday(&start_sort, NULL);
    thrust::device_ptr<float> thr_d(d_edge);
	thrust::device_vector<float>d_sup_vec(thr_d,thr_d+(height*width));
    thrust::sort(d_sup_vec.begin(),d_sup_vec.end());
	float index = (float)(.9*height*width);
	float t_hi = d_sup_vec[index];
    float t_lo =t_hi/2;
    gettimeofday(&end_sort, NULL);

   // printf("index = %d\n", index);
    //printf("thi = %d\n", t_hi);
    //printf("tlo = %d\n", t_lo);

    
    gettimeofday(&start_hys, NULL);
    hysteresis<<<dmGrid, dmBlock>>>(d_edge, height, width, t_hi, t_lo);
    hipDeviceSynchronize();
    gettimeofday(&end_hys, NULL);

    gettimeofday(&start_edge, NULL);
    edgeLinking<<<dmGrid, dmBlock>>>(d_edge, height, width, d_edge );
    hipDeviceSynchronize();
    gettimeofday(&end_edge, NULL);

  


    compute_cornerness<<<dmGrid, dmBlock,sizeof(float)*block_dim*block_dim>>>(d_temp_hor, d_temp_ver, height, width, WINDOW, d_corn);
    hipMemcpy(temp_corn, d_corn,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    float *d_cornerness;
    hipMalloc((void **)&d_cornerness,sizeof(float)*dmGrid.x*dmGrid.y);
    float* max_corner = (float*)malloc(sizeof(float)*dmGrid.x*dmGrid.y);


    corners<<<dmGrid, dmBlock,2*sizeof(float)*block_dim*block_dim>>>(d_corn, height, width, d_cornerness);
    hipMemcpy(max_corner, d_cornerness,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    printf("indices\n");
    for (int e = 0; e < block_dim; e ++){
        printf("%ld, %ld\n", (max_corner[e]/width), ((int)(max_corner[e])%width));

    }

    gettimeofday(&start_memCpydevice, NULL);
    hipMemcpy(temp_hor, d_temp_hor,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_ver, d_temp_ver,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_grad, d_grad,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_mag, d_mag,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    hipMemcpy(temp_edge, d_edge,sizeof(float)*width*height,hipMemcpyDeviceToHost);
    gettimeofday(&end_memCpydevice, NULL);

    /*
    for(l = 0; l < height * width; l++){
       // printf(" %d ", temp_edge[l]);
    }

*/
    
   // printf("End of Func\n");
/*
    for(i = 0; i < height * width; i++){
       printf("%d ", temp_hor[i]);
    }
    printf("\n");
*/
    gettimeofday(&start_edge_write, NULL);
    write_image_template("ansHor", temp_hor, width, height);
    write_image_template("ansVer", temp_ver, width, height);
    write_image_template("ansMag", temp_mag, width, height);
    write_image_template("ansDir", temp_grad, width, height);
    write_image_template("ansEdg", temp_edge, width, height);
    write_image_template("ansCorn", temp_corn, width, height);
    //write_image_template("ansInd", max_corner, width, height);


    gettimeofday(&end_edge_write, NULL);



    hipDeviceSynchronize();
    gettimeofday(&end_total, NULL);


    //printf("Total time :%ld\n", ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec)));
    //printf("Total - File-IO: %ld\n", ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec) - ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)) - ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec))));
    //printf("File IO image read: %ld\n", ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)));
    //printf("File IO image write: %ld\n", ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec)));
    //printf("Host to Device: %ld\n", ((end_memCpy.tv_sec * 1000000 + end_memCpy.tv_usec) - (start_memCpy.tv_sec * 1000000 + start_memCpy.tv_usec)));
    //printf("Horizontal Convolution :%ld\n", ((end_hor.tv_sec * 1000000 + end_hor.tv_usec) - (start_hor.tv_sec * 1000000 + start_hor.tv_usec)));
    //printf("Vertical Convolution: %ld\n", ((end_ver.tv_sec * 1000000 + end_ver.tv_usec) - (start_ver.tv_sec * 1000000 + start_ver.tv_usec)));
    //printf("Magnitude Kernel: %ld\n", ((end_mag.tv_sec * 1000000 + end_mag.tv_usec) - (start_mag.tv_sec * 1000000 + start_mag.tv_usec)));
    //printf("Gradient Kernel: %ld\n", ((end_dir.tv_sec * 1000000 + end_dir.tv_usec) - (start_dir.tv_sec * 1000000 + start_dir.tv_usec)));
    //printf("Thrust Sorting: %ld\n", ((end_sort.tv_sec * 1000000 + end_sort.tv_usec) - (start_sort.tv_sec * 1000000 + start_sort.tv_usec)));
    //printf("Suppression Kernel: %ld\n", ((end_supp.tv_sec * 1000000 + end_supp.tv_usec) - (start_supp.tv_sec * 1000000 + start_supp.tv_usec)));
    //printf("Hysteresis Kernel: %ld\n", ((end_hys.tv_sec * 1000000 + end_hys.tv_usec) - (start_hys.tv_sec * 1000000 + start_hys.tv_usec)));
    //printf("Edge Linking Kernel: %ld\n", ((end_edge.tv_sec * 1000000 + end_edge.tv_usec) - (start_edge.tv_sec * 1000000 + start_edge.tv_usec)));
    //printf("Device to Host: %ld\n", ((end_memCpydevice.tv_sec * 1000000 + end_memCpydevice.tv_usec) - (start_memCpydevice.tv_sec * 1000000 + start_memCpydevice.tv_usec)));



    printf("%ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %ld, %d\n", 
    ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec)),
    ((end_total.tv_sec * 1000000 + end_total.tv_usec) - (start_total.tv_sec * 1000000 + start_total.tv_usec) - ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)) - ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec))), 
    ((end_image_read.tv_sec * 1000000 + end_image_read.tv_usec) - (start_image_read.tv_sec * 1000000 + start_image_read.tv_usec)),
    ((end_edge_write.tv_sec * 1000000 + end_edge_write.tv_usec) - (start_edge_write.tv_sec * 1000000 + start_edge_write.tv_usec)),
    ((end_memCpy.tv_sec * 1000000 + end_memCpy.tv_usec) - (start_memCpy.tv_sec * 1000000 + start_memCpy.tv_usec)),
    ((end_hor.tv_sec * 1000000 + end_hor.tv_usec) - (start_hor.tv_sec * 1000000 + start_hor.tv_usec)),
    ((end_ver.tv_sec * 1000000 + end_ver.tv_usec) - (start_ver.tv_sec * 1000000 + start_ver.tv_usec)),
    ((end_mag.tv_sec * 1000000 + end_mag.tv_usec) - (start_mag.tv_sec * 1000000 + start_mag.tv_usec)),
    ((end_dir.tv_sec * 1000000 + end_dir.tv_usec) - (start_dir.tv_sec * 1000000 + start_dir.tv_usec)),
    ((end_sort.tv_sec * 1000000 + end_sort.tv_usec) - (start_sort.tv_sec * 1000000 + start_sort.tv_usec)),
    ((end_supp.tv_sec * 1000000 + end_supp.tv_usec) - (start_supp.tv_sec * 1000000 + start_supp.tv_usec)),
    ((end_hys.tv_sec * 1000000 + end_hys.tv_usec) - (start_hys.tv_sec * 1000000 + start_hys.tv_usec)),
    ((end_edge.tv_sec * 1000000 + end_edge.tv_usec) - (start_edge.tv_sec * 1000000 + start_edge.tv_usec)),
    ((end_memCpydevice.tv_sec * 1000000 + end_memCpydevice.tv_usec) - (start_memCpydevice.tv_sec * 1000000 + start_memCpydevice.tv_usec)),
    block_dim);

    free(temp_hor);
    free(temp_ver);
    hipFree(d_image);
    hipFree(d_temp_hor);
    hipFree(d_temp_ver);
    hipFree(d_kernel);
    hipFree(d_dkernel);
    hipFree(d_grad);
    hipFree(d_mag);
    free(temp_grad);
    free(temp_mag);
    hipFree(d_edge);
    free(temp_edge);
    hipFree(d_corn);
    free(temp_corn);


    return 0;




}
